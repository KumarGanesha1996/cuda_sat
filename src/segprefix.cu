
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>
#include <vector>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define BATCH_SIZE 4096
#define WARPS_NB 10
#define abs8(n) ((n) & 0x7fu)
#define abs32(n) ((n) & 0x7fffffffu)

struct clause {
	/* Field 'flags':
	 *   0x01u - value of literal l[0] (taking into account the sign)
	 *   0x02u - value of literal l[1] (taking into account the sign)
	 *   0x04u - value of literal l[2] (taking into account the sign)
	 *   0x08u - literal l[0] was assigned a value
	 *   0x10u - literal l[1] was assigned a value
	 *   0x20u - literal l[2] was assigned a value
	 * Satisfied if: (flags & 0x07u) != 0x00u
	 * Has literals: (flags & 0x38u) != 0x38u
	 * Invalid if:   (flags & 0x3fu) == 0x2au
	 */
	int8_t l[3];
	uint8_t flags;
#define c_sat(c) (((c).flags & 0x07u) != 0x00u)
#define c_has(c) (((c).flags & 0x38u) != 0x38u)
#define c_inv(c) (((c).flags & 0x3fu) == 0x38u)
};

/************************* PREPROCESS ****************************/

__global__ void preprocess(clause *d_f1, unsigned int *d_v, int r) {
	int warp_id = WARPS_NB * blockIdx.x + (threadIdx.x >> 5); // check
	int lane_id = threadIdx.x & 31;
	clause *formula = d_f1 + warp_id * r;
	unsigned int *valid = d_v + warp_id; // check

	// dodac ifa jezeli jestesmy warpem niezerowym? on juz ma te dane przeciez?
	// o nie... musi byc osobna tablica... co jak warp 0 juz zabierze sie do roboty?
	for(int i = threadIdx.x & 31; i < r; ++i) {
		formula[i] = d_f1[i];
	}

	int number = warp_id;

	while(number) { // check
		int tmp = number / 3;
		int branch_id = number - 3 * tmp;
		number = tmp;
		clause fc;
		bool fc_found = false;
		unsigned int mask1 = 0xffffffffu; // check

		for(int i = lane_id; true; i += 32) {
			mask1 = __ballot_sync(mask1, i < r); // check for second loop

			if(i >= r) {
				break;
			}

			clause cl = formula[i];

			if(!fc_found) {
				int has_literals = c_has(cl); // check and/or improve
				int mask2 = __ballot_sync(mask1, has_literals); // check if it is OK

				if(!mask2) {
					continue;
				}

				fc_found = true;
				int *ptr_cl = (int *) &cl;
				int src_lane_id = __ffs(mask2) - 1;
				tmp = __shfl_sync(mask1, *ptr_cl, src_lane_id);
				fc = *((clause *) &tmp);

				if(!(fc.flags & (0x08u << branch_id))) {
					if(lane_id == 0) {
						*valid = 0;
					}

					return;
				}
			}

			for(int l = 0; l < 3; ++l) {
				for(int x = 0; x < branch_id; ++x) {
					if(!(cl.flags & (0x08u << l)) && abs8(cl.l[l]) == abs8(fc.l[x])) {
						cl.flags |= (0x08u + (fc.l[x] < 0)) << l;
					}
				}

				if(cl.l[l] == fc.l[branch_id]) {
					cl.flags |= (0x08u + (fc.l[branch_id] > 0)) << l;
				}
			}

			if(__any_sync(0xffffffffu, c_inv(cl))) {
				if(lane_id == 0) {
					*valid = 0;
				}

				return;
			}

			formula[i] = cl;
		}

		if(!fc_found) {
			// whole formula satisfied! I think...
			return;
		}
	}
}

/************************* SAT_KERNEL ****************************/

/* Triples the number of formulas in a batch and marks invalid/missing ones
 * 
 * d_f - array of formulas and a free space for new formulas
 * d_v - array of flags indicating whether a formula is valid or not
 * k - number of formulas to triple
 * r - total number of clauses
 */
__global__ void sat_kernel(clause *d_f1, clause *d_f2, unsigned int *d_v, int k, int r) {
	int lane_id = threadIdx.x & 31;
	int warp_id = WARPS_NB * blockIdx.x + (threadIdx.x >> 5);
	int formula_id = warp_id / 3;
	int branch_id = warp_id - 3 * formula_id;
	unsigned int *valid = d_v + k * branch_id + formula_id;
	clause *formula = d_f1 + formula_id * r;
	clause *destination = d_f2 + (branch_id * k + formula_id) * r;
	clause fc = formula[0]; // this might be slow, use __shfl_sync()?

	// check
	if(!(fc.flags & (0x08u << branch_id))) {
		if(lane_id == 0) {
			*valid = 0;
		}

		return;
	}

	for(int i = lane_id; i < r; i += 32) {
		clause cl = formula[i];

		if(c_sat(cl)) { // sprawdzic czy jest dobrze: jak jest nullowalna, to uciekaj
			break;
		}

		for(int l = 0; l < 3; ++l) {
			for(int x = 0; x < branch_id; ++x) {
				if(!(cl.flags & (0x08u << l)) && abs8(cl.l[l]) == abs8(fc.l[x])) {
					cl.flags |= (0x08u + (fc.l[x] < 0)) << l;
				}
			}

			if(cl.l[l] == fc.l[branch_id]) {
				cl.flags |= (0x08u + (fc.l[branch_id] > 0)) << l;
			}
		}

		// check
		if(__any_sync(0xffffffffu, c_inv(cl))) {
			if(lane_id == 0) {
				*valid = 0;
			}

			return;
		}

		destination[i] = cl;
	}
}

/*************************** SCAN_1D *****************************/

__device__ unsigned int id = 0;
__device__ unsigned int d_p[32];
__device__ unsigned int valid_f;

__inline__ __device__ unsigned int warp_scan(unsigned int v) {
	int lane_id = threadIdx.x & 31;

	for(int i = 1; i < 32; i <<= 1) {
		int _v = __shfl_up_sync(0xffffffffu, v, i);

		if(lane_id >= i) {
			v += abs32(_v);
		}
	}

	return v;
}

__global__ void scan_1d(unsigned int *d_v, int k, int range_parts, int range) {
	__shared__ unsigned int partials[33];
	__shared__ unsigned int prev;
	int tid = blockIdx.x * range + threadIdx.x;
	int warp_id = threadIdx.x >> 5;
	int lane_id = threadIdx.x & 31;

	if(tid == 0) {
		partials[0] = 0;
		prev = 0;
	}

	__syncthreads();

	for(int i = 0; i < range_parts && tid < k; tid += 1024) {
		unsigned int v = warp_scan(d_v[tid]);

		if(lane_id == 31) {
			partials[warp_id + 1] = v;
		}

		__syncthreads();

		if(warp_id == 0) {
			partials[lane_id] = warp_scan(partials[lane_id]);
		}

		__syncthreads();

		d_v[tid] = v + prev;

		__syncthreads();

		if((tid & 1023) == 1023) {
			prev = abs32(v);
		}
	}

	if((tid & 1023) == 1023) {
		d_p[blockIdx.x] = prev;
		__threadfence();

		if(atomicAdd(&id, 1) == gridDim.x - 1) {
			id = 0;
			d_p[lane_id] = warp_scan(d_p[lane_id]);
		}
	}
}

__global__ void propagate_1d(unsigned int *d_v, int k, int range_parts, int range) {
	__shared__ int prev;
	int tid = (blockIdx.x + 1) * range + threadIdx.x;

	if(threadIdx.x == 0) {
		prev = d_p[blockIdx.x];
	}

	__syncthreads();

	unsigned int v;

	for(int i = 0; i < range_parts && tid < k; tid += 1024) {
		v = d_v[tid] += prev;
	}

	if(tid == k + 1023) {
		valid_f = v;
	}
}

/************************** SCATTER_1D ***************************/

__global__ void scatter_1d(clause *d_f1, clause *d_f2, int *d_v, int k, int r) {
	int warp_id = (blockIdx.x << 5) + (threadIdx.x >> 5);
	unsigned int p = d_v[warp_id];
	unsigned int valid = p & 0x80000000u;
	clause *formula = d_f2 + warp_id * r;
	clause *destination = d_f1 + (valid ? abs32(p) - 1 : valid_f + warp_id - abs32(p)) * r;

	for(int i = threadIdx.x & 31; i < r; i += 32) {
		destination[i] = formula[i];
	}
}

/*************************** SCAN_2D *****************************/

__inline__ __device__ unsigned int warp_scan(unsigned int v, int reminder, int lane_id) {
	for(int i = 1; i < 32; i <<= 1) {
		int _v = __shfl_up_sync(0xffffffffu, v, i);

		if(lane_id >= i && i <= reminder) { // chyba dobrze
			v += abs32(_v);
		}
	}

	return v;
}

__global__ void scan_2d(clause *d_f1, int *d_v, int k, int r, int range_parts, int range) {
	__shared__ int partials[33];
	__shared__ int prev;
	int tid = blockIdx.x * range + threadIdx.x;
	int warp_id = threadIdx.x >> 5;
	int lane_id = threadIdx.x & 31;
	int range_start = tid;

	if(tid == 0) {
		partials[0] = 0;
		prev = 0;
	}

	__syncthreads();

	for(int i = 0; i < range_parts && tid < k; tid += 1024) {
		int reminder = tid % r; // da sie ifami, ale remainder zwiekszam o 1024%r if(remainder >= r) { remainder -= r; }
	clause cl = d_f1[tid];
	unsigned int satisfied = c_sat(cl) ? 0 : 0x80000001u;
	unsigned int v = warp_scan(satisfied, reminder, lane_id);

	if(lane_id == 31) {
		partials[warp_id + 1] = v;
	}

	__syncthreads();

	if(warp_id == 0) {
		partials[lane_id] = warp_scan(partials[lane_id]);
	}

	__syncthreads();

	if(tid - range_start <= reminder) { // chyba dobrze
		d_v[tid] = v + prev;
	}

	__syncthreads();

	if((tid & 1023) == 1023) {
		prev = abs32(v);
	}
	}

	if((tid & 1023) == 1023) {
		d_p[blockIdx.x] = prev;
		__threadfence();

		if(atomicAdd(&id, 1) == gridDim.x - 1) {
			id = 0;
			d_p[lane_id] = warp_scan(d_p[lane_id]);
		}
	}
}

// NIE MA 2D_PROPAGATE

/************************** SCATTER_2D ***************************/

__global__ void scatter_2d(clause *d_f1, clause *d_f2, int *d_v, int r) {
	int warp_id = (blockIdx.x << 5) + (threadIdx.x >> 5);
	int shift = warp_id * r;
	int *position = d_v + shift;
	clause *formula = d_f1 + shift;
	clause *destination = d_f2 + shift;

	for(int i = threadIdx.x & 31; i < r; i += 32) {
		int p = position[i]; // check!
		unsigned int satisfied = p & 0x80000000u; // check!
		destination[satisfied ? p - 1 : valid_f + warp_id - p] = formula[i];
	}
}

/**************************** SWAP *******************************/

__managed__ bool formula_satisfied = false; // pewnie wyzej umiescic i sprawdzic kiedys???

void swap() {

}

/************************** PIPELINE *****************************/

void pipeline(std::vector<clause> &formulas, int n, int r, int s) {
	// in main
}

/************************ EXTRACT_VARS ***************************/

// from a formula, extracts variables

void extract_vars(clause *formula, int r, std::vector<bool> &assignment) {
	for(int i = 0; i < r; ++i) {
		for(int j = 0; j < 3; ++j) {
			int8_t var = formula[i].l[j];
			bool val = formula[i].flags & (0x01u << j);
			bool set = formula[i].flags & (0x08u << j);

			if(set) {
				assignment[abs8(var)] = (var < 0) ^ val; 
			}
		}
	}
}

void print_formula(clause *formula, int r) {
	for(int i = 0; i < r; ++i) {
		uint8_t *ptr = (uint8_t *) &formula[i];

		for(int j = 0; j < sizeof(clause); ++j) {
			for(int k = 0; k < 8; ++k) {
				printf("%d", (ptr[j] >> 7-k) & 1);
			}

			printf(" ");
		}

		printf("\n");
	}

	printf("\n");
}

/**************************** MAIN *******************************/

int main() {
	int n, r;
	scanf("%d %d", &n, &r);
	// int nb_of_formulas = 1;
	int s = 1;

	while(3 * s <= BATCH_SIZE) {
		s *= 3;
	}

	std::vector<clause> formulas(BATCH_SIZE * r);

	for(int i = 0; i < r; ++i) {
		int j = 0;

		while(j < 3) {
			int var;
			scanf("%d", &var);

			if(var == 0) {
				break;
			}

			formulas[i].l[j] = (int8_t) var;
			++j;
		}

		while(j < 3) {
			formulas[i].flags &= 0x8u << j;
			++j;
		}
	}

	print_formula(formulas.data(), r);

	/*
	   int parts = (n + 32 * 1024 - 1) / (32 * 1024);
	   int range = parts * 1024;
	   int blocks = (n + range - 1)/range;
	   int *d_values, *d_groups, *d_groups_original;
	 */

	/*
	   clause *d_f1;
	   clause *d_f2;
	   unsigned int *d_v;
	   cudaMallocHost(&d_f1, BATCH_SIZE * r * sizeof(clause));
	   cudaMallocHost(&d_f2, BATCH_SIZE * r * sizeof(clause));
	   cudaMallocHost(&d_v, BATCH_SIZE * sizeof(unsigned int));
	   cudaMemcpyAsync(&d_f1, formulas.data(), r * sizeof(clause), cudaMemcpyDefault);

	   preprocess<<<0, 0>>>(d_f1, d_f2, d_v, s, r);

	   while(true) {
	   scan_1d<<<0, 0>>>(d_v, s, range_parts, range); // check nb_of_formulas
	// jezeli blokow jest 1, to nie rob propagate
	scatter_1d<<<0, 1024>>>(d_f1);
	scan_1d<<<0, 1024>>>(d_f1, d_v, range_parts, range);
	scatter_1d<<<0, 1024>>>(d_f1, d_f2, d_v, r);
	swap();
	sat_kernel<<<0, 0>>>(d_f1, d_f2, d_v, BATCH_SIZE, r);
	}

	cudaFree(d_f1);
	cudaFree(d_f2);
	cudaFree(d_v);
	 */
	/*
	   fill_groups<<<(m + 1023)/1024, 1024>>>(d_groups, d_groups_original, m);
	   block_prefix_scan<<<blocks, 1024>>>(d_values, d_groups, n, parts, range);
	 */
}

